#include "hip/hip_runtime.h"
/****
 * Copyright (c) 2011-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ****/

#include <string.h>
#include <stdio.h>
#include <assert.h>
#include <unistd.h>
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <mp.h>
#include <mp/device.cuh>

#include "test_utils.h"

#define MAX_SIZE 64*1024
// NOTE: cannot iterate too much before exhausting resources, like CQs and WQs.
#define ITER_COUNT_SMALL 20
#define ITER_COUNT_LARGE 1
#define WINDOW_SIZE 64 

int comm_size, my_rank, peer;

struct comm_descs {
    enum { max_n_descs = ITER_COUNT_SMALL };
    mp::mlx5::send_desc_t tx[max_n_descs];
    mp::mlx5::wait_desc_t tx_wait[max_n_descs];
    mp::mlx5::wait_desc_t rx_wait[max_n_descs];
};

__global__ void exchange_kernel(int my_rank, comm_descs descs, int iter_count)
{
    int i;
    assert(gridDim.x == 1);

    //if (threadIdx.x == 0) printf("iter_count=%d\n", iter_count);

    for (i=0; i<iter_count; ++i) {
        if (!my_rank) {
            if (0 == threadIdx.x) {
                //printf("i=%d send+recv\n", i);
                // make sure NIC can fetch coherent data
                __threadfence();
                mp::device::mlx5::send(descs.tx[i]);
                mp::device::mlx5::wait(descs.tx_wait[i]);
                mp::device::mlx5::signal(descs.tx_wait[i]);
                mp::device::mlx5::wait(descs.rx_wait[i]);
                mp::device::mlx5::signal(descs.rx_wait[i]);
            }
            __syncthreads();
        } else {
            if (0 == threadIdx.x) {
                //printf("i=%d recv+send\n", i);
                // make sure NIC can fetch coherent data
                __threadfence();
                mp::device::mlx5::wait(descs.rx_wait[i]);
                mp::device::mlx5::signal(descs.rx_wait[i]);
                mp::device::mlx5::send(descs.tx[i]);
                mp::device::mlx5::wait(descs.tx_wait[i]);
                mp::device::mlx5::signal(descs.tx_wait[i]);
            }
            __syncthreads();
        }
    }
}

int launch_exchange_kernel(int my_rank, comm_descs &descs, int iter_count, hipStream_t stream)
{
    exchange_kernel<<<1,16,0,stream>>>(my_rank, descs, iter_count);
    CUDA_CHECK(hipGetLastError());
    return 0;
}

int sr_exchange (MPI_Comm comm, int size, int iter_count, int validate)
{
    int j;
    size_t buf_size; 
    hipStream_t stream;

    /*application and pack buffers*/
    void *buf = NULL, *sbuf_d = NULL, *rbuf_d = NULL;

    /*mp specific objects*/
    mp_request_t *sreq = NULL;
    mp_request_t *rreq = NULL;
    mp_reg_t sreg, rreg; 

    buf_size = size*iter_count;

    /*allocating requests*/
    sreq = (mp_request_t *) malloc(iter_count*sizeof(mp_request_t));
    rreq = (mp_request_t *) malloc(iter_count*sizeof(mp_request_t));

    hipHostMalloc(&buf, buf_size);
    memset(buf, 0, buf_size); 

    CUDA_CHECK(hipMalloc((void **)&sbuf_d, buf_size));
    CUDA_CHECK(hipMemset(sbuf_d, 0, buf_size)); 

    CUDA_CHECK(hipMalloc((void **)&rbuf_d, buf_size));
    CUDA_CHECK(hipMemset(rbuf_d, 0, buf_size)); 
 
    CUDA_CHECK(hipStreamCreate(&stream));	

    MP_CHECK(mp_register(sbuf_d, buf_size, &sreg, 0));
    MP_CHECK(mp_register(rbuf_d, buf_size, &rreg, 0));

    if (validate) {
        CUDA_CHECK(hipMemset(sbuf_d, (my_rank + 1), buf_size));
        CUDA_CHECK(hipMemset(rbuf_d, 0, buf_size));
    }

    comm_descs descs;

    for (j = 0; j < iter_count; j++) {
        assert(j < comm_descs::max_n_descs);
        // note: the ordering is not important here, no risk of deadlocks
        if (!my_rank) { 
            MP_CHECK(mp_send_prepare((void *)((uintptr_t)sbuf_d + size*j), size, peer, &sreg, &sreq[j]));
            MP_CHECK(mp::mlx5::get_descriptors(&descs.tx[j],      &sreq[j]));
            MP_CHECK(mp::mlx5::get_descriptors(&descs.tx_wait[j], &sreq[j]));

            MP_CHECK(mp_irecv ((void *)((uintptr_t)rbuf_d + size*j), size, peer, &rreg, &rreq[j]));
            MP_CHECK(mp::mlx5::get_descriptors(&descs.rx_wait[j], &rreq[j]));
        } else {
            MP_CHECK(mp_irecv ((void *)((uintptr_t)rbuf_d + size*j), size, peer, &rreg, &rreq[j]));
            MP_CHECK(mp::mlx5::get_descriptors(&descs.rx_wait[j], &rreq[j]));

            MP_CHECK(mp_send_prepare((void *)((uintptr_t)sbuf_d + size*j), size, peer, &sreg, &sreq[j]));
            MP_CHECK(mp::mlx5::get_descriptors(&descs.tx[j],      &sreq[j]));
            MP_CHECK(mp::mlx5::get_descriptors(&descs.tx_wait[j], &sreq[j]));
        }
    }
    //printf("launching kernel iter_count=%d\n", iter_count);
    launch_exchange_kernel(my_rank, descs, iter_count, stream);
    //CUDA_CHECK(hipStreamSynchronize(stream));
    //printf("waiting for recv reqs\n");
    MP_CHECK(mp_wait_all(iter_count, rreq));
    //printf("waiting for send reqs\n");
    MP_CHECK(mp_wait_all(iter_count, sreq));
    // all ops in the stream should have been completed 
    usleep(1000);
    CUDA_CHECK(hipStreamQuery(stream));
    MPI_CHECK(MPI_Barrier(comm));

    if (validate && my_rank) {
        CUDA_CHECK(hipMemcpy(buf, rbuf_d, buf_size, hipMemcpyDefault));
        char *value = (char*)buf;
        char expected = (char) (peer + 1);
        for (j=0; j<(iter_count*size); j++) {
             if (value[j] != (peer + 1)) {
                fprintf(stderr, "validation check failed index: %d expected: %d actual: %d \n", j, expected, value[j]);
                 exit(-1);
             }
        }
    }
    MPI_CHECK(MPI_Barrier(comm));
    CUDA_CHECK(hipDeviceSynchronize());
    mp_deregister(&sreg);
    mp_deregister(&rreg);
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(sbuf_d));
    CUDA_CHECK(hipFree(rbuf_d));
    hipHostFree(buf);
    free(sreq);
    free(rreq);

    return 0;
}

int main (int c, char *v[])
{
    int iter_count, size;
    int validate = 1;

    MPI_CHECK(MPI_Init(&c, &v));
    MPI_CHECK(MPI_Comm_size(MPI_COMM_WORLD, &comm_size));
    MPI_CHECK(MPI_Comm_rank(MPI_COMM_WORLD, &my_rank));

    if (comm_size != 2) { 
	fprintf(stderr, "this test requires exactly two processes \n");
        exit(-1);
    }

    if (gpu_init(-1)) {
        fprintf(stderr, "got error while initializing GPU\n");
        MPI_Abort(MPI_COMM_WORLD, -1);
    }

    peer = !my_rank;
    //Need to set CUDA_VISIBLE_DEVICES
    MP_CHECK(mp_init(MPI_COMM_WORLD, &peer, 1, MP_INIT_DEFAULT, 0));

    iter_count = ITER_COUNT_SMALL;

    for (size=1; size<=MAX_SIZE; size*=2) 
    {
        if (size > 1024) {
            iter_count = ITER_COUNT_LARGE;
        }

        sr_exchange(MPI_COMM_WORLD, size, iter_count, validate);

        if (!my_rank) fprintf(stdout, "# SendRecv test passed validation with message size: %d \n", size);
    }

    mp_finalize();
    MPI_CHECK(MPI_Barrier(MPI_COMM_WORLD));
    MPI_CHECK(MPI_Finalize());
    return 0;
}
