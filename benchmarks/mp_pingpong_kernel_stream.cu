#include "hip/hip_runtime.h"
/****
 * Copyright (c) 2011-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ****/

#include "mpi.h"
#include "mp.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include "assert.h"
#include <limits.h>
#include <sys/types.h>
#include <unistd.h>
#include "prof.h"
#include "hip/hip_runtime_api.h"

#define CUDA_CHECK(stmt)                                \
do {                                                    \
    hipError_t result = (stmt);                        \
    if (hipSuccess != result) {                        \
        fprintf(stderr, "[%s:%d] cuda failed with %s \n",   \
         __FILE__, __LINE__,hipGetErrorString(result));\
        exit(-1);                                       \
    }                                                   \
    assert(hipSuccess == result);                      \
} while (0)

#define MP_CHECK(stmt)                                  \
do {                                                    \
    int result = (stmt);                                \
    if (0 != result) {                                  \
        fprintf(stderr, "[%s:%d] mp call failed \n",    \
         __FILE__, __LINE__);                           \
        MPI_Abort(MPI_COMM_WORLD, -1);                  \
    }                                                   \
    assert(0 == result);                                \
} while (0)

int enable_debug_prints = 0;
#define mp_dbg_msg(FMT, ARGS...)  do                                    \
{                                                                       \
    if (enable_debug_prints)  {                                              \
        fprintf(stderr, "[%d] [%d] MP DBG  %s() " FMT, getpid(),  my_rank, __FUNCTION__ , ## ARGS); \
        fflush(stderr);                                                 \
    }                                                                   \
} while(0)

#define MAX_SIZE 4096 //128*1024 
//64*1024
#define ITER_COUNT_SMALL (2*1024)
#define ITER_COUNT_LARGE 256


struct prof prof_normal;
struct prof prof_async;
int prof_start = 0;
int prof_idx = 0;

int comm_size, my_rank, peer;
int steps_per_batch = 16, batches_inflight = 4;
int enable_async = 1;

__device__ int counter;
__device__ int clockrate;

__global__ void dummy_kernel(double time)
{
    long long int start, stop;
    double usec;

    start = clock64();
    do {
        stop = clock64();
	usec = ((double)(stop-start)*1000)/((double)clockrate); 
	counter = usec;
    } while(usec < time);
}

/*application and pack buffers*/
void *buf = NULL, *sbuf_d = NULL, *rbuf_d = NULL;
int req_max_inflight = 0, rreq_max_inflight = 0, prepost_depth = 0;
hipStream_t stream;
size_t buf_size;

int gpu_id = -1;
int wait_key = 0;

/*mp specific objects*/
mp_request_t *sreq = NULL;
mp_request_t *rreq = NULL;
mp_reg_t sreg, rreg;
double time_start, time_stop;

double sr_exchange (MPI_Comm comm, int size, int iter_count, int validate, double kernel_time, int use_async, struct prof *prof)
{
    int i, j;
    double latency;
    double time_start, time_stop;
    int req_idx = 0, rreq_idx = 0, complete_req_idx = 0, complete_rreq_idx = 0;
    int req_inflight = 0, rreq_inflight = 0;

    mp_dbg_msg("size=%d iter_count=%d kernel_time=%f use_async=%d\n", size, iter_count, kernel_time, use_async);

    if (validate) {
        mp_dbg_msg("initializing the buffer \n");
        CUDA_CHECK(hipMemset(sbuf_d, (size + 1)%CHAR_MAX, buf_size));
        CUDA_CHECK(hipMemset(rbuf_d, 0, buf_size));
        CUDA_CHECK(hipDeviceSynchronize());
    }

    time_start = MPI_Wtime();

    for (j=0; j<prepost_depth; j++) {
        mp_dbg_msg("posted recv request: %d \n", rreq_idx);
        MP_CHECK(mp_irecv ((void *)((uintptr_t)rbuf_d + size*j), size, peer, &rreg, &rreq[rreq_idx]));
        rreq_idx = (rreq_idx + 1)%rreq_max_inflight;
        rreq_inflight++;
    }

    uint32_t wait_flag;
    if (use_async && wait_key && (1 == my_rank)) {
        fprintf(stdout, "[%d] waiting enabled, inserting a wait32_on_stream\n", my_rank); fflush(stdout);
        ACCESS_ONCE(wait_flag) = 0;
        MP_CHECK(mp_wait32_on_stream(&wait_flag, 1, MP_WAIT_GEQ, stream));
    }


    prof_idx = 0;
    for (j = 0; j < iter_count; j++) {
	mp_dbg_msg("iteration :%d \n", j);

        if (!my_rank) {
            if (prof) PROF(prof, prof_idx++);
            req_idx = j%rreq_max_inflight;
            if (!use_async) {
                MP_CHECK(mp_wait(&rreq[req_idx]));
            } else {
                MP_CHECK(mp_wait_on_stream(&rreq[req_idx], stream));
            }

            if (prof) PROF(prof, prof_idx++);

            if (kernel_time > 0) {
                dummy_kernel <<<1, 1, 0, stream>>> (kernel_time);
                if (!use_async) {
                    CUDA_CHECK(hipStreamSynchronize(stream));
                }
            }

            if (prof) PROF(prof, prof_idx++);

            req_idx = j%req_max_inflight;
            if (!use_async) {
                MP_CHECK(mp_isend ((void *)((uintptr_t)sbuf_d + size*j), size, peer, &sreg, &sreq[req_idx]));
            } else {
                MP_CHECK(mp_isend_on_stream ((void *)((uintptr_t)sbuf_d + size*j), size, peer, &sreg, &sreq[req_idx], stream));
            mp_dbg_msg("posted send request: %d \n", req_idx);
            }
        } else {
            req_idx = j%req_max_inflight;

            if (!use_async) {
                MP_CHECK(mp_isend ((void *)((uintptr_t)sbuf_d + size*j), size, peer, &sreg, &sreq[req_idx]));
            } else {
                MP_CHECK(mp_isend_on_stream ((void *)((uintptr_t)sbuf_d + size*j), size, peer, &sreg, &sreq[req_idx], stream));
            }
            mp_dbg_msg("posted send request: %d\n", req_idx);

            req_idx = j%rreq_max_inflight;
            if (!use_async) {
                MP_CHECK(mp_wait(&rreq[req_idx]));
            } else {
                MP_CHECK(mp_wait_on_stream(&rreq[req_idx], stream));
            }

            if (kernel_time > 0) {
                dummy_kernel <<<1, 1, 0, stream>>> (kernel_time);
                if (!use_async) {
                    CUDA_CHECK(hipStreamSynchronize(stream));
                }
            }
        }

        req_inflight++;
        mp_dbg_msg("requests inflight: %d \n", req_inflight);

        if (!my_rank && prof)  PROF(prof, prof_idx++);

        if ((j + prepost_depth) < iter_count) {
            mp_dbg_msg("posted recv request: %d\n", rreq_idx);
            int buf_idx = (j + prepost_depth);
            MP_CHECK(mp_irecv ((void *)((uintptr_t)rbuf_d + size*buf_idx), size, peer, &rreg, &rreq[rreq_idx]));
            rreq_idx = (rreq_idx + 1)%rreq_max_inflight;
            rreq_inflight++;
        }

        if (!my_rank && prof)  PROF(prof, prof_idx++);

        if (use_async && wait_key && (1 == my_rank)) {
            fprintf(stdout, "[%d] sleeping 15s\n", my_rank);
            sleep(15);
            ACCESS_ONCE(wait_flag) = 1;
            // disabling wait_key for subsequent calls
            wait_key = 0;
            fprintf(stdout, "[%d] sleeping 20us to let previous batches to run\n", my_rank);
            usleep(20);
            fprintf(stdout, "[%d] resuming...\n", my_rank);
            fflush(stdout);
        }

        /*synchronize on oldest batch*/
        if (req_inflight == req_max_inflight) {
	    if (use_async) { 
	        for (i=0; i<steps_per_batch; i++) {
	            mp_dbg_msg("waiting on recv request: %d\n", complete_rreq_idx);
                    MP_CHECK(mp_wait(&rreq[complete_rreq_idx]));
	            mp_dbg_msg("completed recv request: %d\n", complete_rreq_idx);
                    complete_rreq_idx = (complete_rreq_idx + 1)%rreq_max_inflight;
                    rreq_inflight--;
                } 
                mp_dbg_msg("after waiting on recv, rreq_inflight: %d \n", rreq_inflight);
	    }

	    for (i=0; i<steps_per_batch; i++) {
		mp_dbg_msg("waiting on send request: %d \n", complete_req_idx);
                MP_CHECK(mp_wait(&sreq[complete_req_idx]));
		mp_dbg_msg("completed send request: %d \n", complete_req_idx);
                complete_req_idx = (complete_req_idx + 1)%req_max_inflight;
                req_inflight--;
            }
	    mp_dbg_msg("after waiting on send, req_inflight: %d \n", req_inflight);

        }

        if (j == (iter_count - 1)) {
	    /*ideally, there should be validation here*/
	    if (use_async) {
                while (rreq_inflight > 0) {
                    mp_wait(&rreq[complete_rreq_idx]);
                    mp_dbg_msg("completed recv request: %d \n", complete_rreq_idx);
                    complete_rreq_idx = (complete_rreq_idx + 1)%rreq_max_inflight;
                    rreq_inflight--;
                }
	    }

            while (req_inflight > 0) {
                mp_wait(&sreq[complete_req_idx]);
                mp_dbg_msg("completed send request: %d \n", complete_req_idx);
                complete_req_idx = (complete_req_idx + 1)%req_max_inflight;
                req_inflight--;
            }
        }

        if (!my_rank && prof)  {
            PROF(prof, prof_idx++);
            prof_update(prof);
            prof_idx = 0;
        }
    }

    // TODO: move validate after timing
    if (validate) {
        CUDA_CHECK(hipMemcpy((void *)((uintptr_t)buf), (void *)((uintptr_t)rbuf_d), 
        	buf_size, hipMemcpyDefault));
	//CUDA_CHECK(hipDeviceSynchronize());

        char *value = (char *)((uintptr_t)buf);
        for (i=0; i<buf_size; i++) {
             if (value[i] != (size + 1)%CHAR_MAX) {
                 mp_dbg_msg("validation check failed index: %d expected: %d actual: %d \n", 
                            i, (size + 1)%CHAR_MAX, value[i]);
                 exit(-1);
             }
        }
    }

    MPI_Barrier(comm);

    time_stop = MPI_Wtime();
    latency = (((time_stop - time_start)*1e6)/(iter_count*2));

    CUDA_CHECK(hipDeviceSynchronize());

    return latency;
}

int main (int argc, char *argv[])
{
    int iter_count = 0, size = 0, dev_count = 0, local_rank = 0, dev_id = 0;
    int kernel_time = 20;
    int comm_comp_ratio = 0;
    int validate = 0;
    int max_size = MAX_SIZE; 

    char *value = getenv("ENABLE_VALIDATION");
    if (value != NULL) {
        validate = atoi(value);
    }

    value = getenv("ENABLE_DEBUG_MSG");
    if (value != NULL) {
        enable_debug_prints = atoi(value);
    }

    value = getenv("KERNEL_TIME");
    if (value != NULL) {
        kernel_time = atoi(value);
    }

    value = getenv("COMM_COMP_RATIO");
    if (value != NULL) {
        comm_comp_ratio = atoi(value);
    }

    size = 1;
    value = getenv("SIZE");
    if (value != NULL && atoi(value)) {
        size = atoi(value);
    }

    value = getenv("MAX_SIZE");
    if (value != NULL && atoi(value)) {
        max_size = atoi(value);
    }

    int event_async = 0;
    value = getenv("MP_EVENT_ASYNC");
    if (value != NULL) {
        event_async = atoi(value);
    }

    while(1) {
        int c;
        c = getopt(argc, argv, "d:W:s:");
        if (c == -1)
            break;

        switch(c) {
        case 'd':
            gpu_id = strtol(optarg, NULL, 0);
            break;
        case 's':
            size = strtol(optarg, NULL, 0);
            printf("size=%d\n", size);
            break;
        case 'W':
            wait_key = strtol(optarg, NULL, 0);
            printf("wait_key=%d\n", wait_key);
            break;
        default:
            printf("ERROR: invalid option\n");
            exit(EXIT_FAILURE);
        }
    }

    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &comm_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &my_rank);

    if (comm_size != 2) {
        fprintf(stderr, "this test requires exactly two processes \n");
        exit(-1);
    }

    CUDA_CHECK(hipGetDeviceCount(&dev_count));
    if (dev_count <= 0) {
        fprintf(stderr, "no CUDA devices found \n");
        exit(-1);
    }

    if (gpu_id >= 0) {
        local_rank = gpu_id;
    } else if (getenv("USE_GPU")) {
        local_rank = atoi(getenv("USE_GPU"));
    } else if (getenv("MV2_COMM_WORLD_LOCAL_RANK") != NULL) {
        local_rank = atoi(getenv("MV2_COMM_WORLD_LOCAL_RANK"));
    } else if (getenv("OMPI_COMM_WORLD_LOCAL_RANK") != NULL) {
        local_rank = atoi(getenv("OMPI_COMM_WORLD_LOCAL_RANK"));
    }

    dev_id = local_rank%dev_count;
    fprintf(stdout, "[%d] local_rank: %d dev_count: %d using GPU device: %d\n", my_rank, local_rank, dev_count, dev_id);
    fprintf(stdout, "[%d] validate=%d event_async=%d\n", my_rank, validate, event_async);
    CUDA_CHECK(hipSetDevice(dev_id));
    CUDA_CHECK(hipFree(0));

    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, dev_id));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(clockrate), (void *)&prop.clockRate, sizeof(int), 0, hipMemcpyHostToDevice));
    fprintf(stdout, "[%d] GPU name=%s\n", my_rank, prop.name);

    peer = !my_rank;
    MP_CHECK(mp_init (MPI_COMM_WORLD, &peer, 1, MP_INIT_DEFAULT, dev_id));

    iter_count = ITER_COUNT_SMALL;
    if (!my_rank) {
        fprintf(stdout, "steps_per_batch: %d batches_inflight: %d \n",
                steps_per_batch, batches_inflight);
        fprintf(stdout, "WARNING: dumping half round-trip latency!!!\n");
    }

    prepost_depth = (steps_per_batch < iter_count) ? steps_per_batch : iter_count;
    req_max_inflight = steps_per_batch*batches_inflight;
    rreq_max_inflight = (steps_per_batch*batches_inflight + prepost_depth);

    /*allocating requests*/
    sreq = (mp_request_t *) malloc(req_max_inflight*sizeof(mp_request_t));
    rreq = (mp_request_t *) malloc(rreq_max_inflight*sizeof(mp_request_t));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    if (!my_rank) fprintf(stdout, "%10s\t  %10s\t    %10s\t    %10s %10s\t    %10s\n", "Size", "KernelTime", "No-async", "No-async+Kernel", "Async", "Async+Kernel");
    if (size != 1) size = max_size = size;
    for (; size<=max_size; size*=2)
    {
        double latency;
        const char *tags = "kernel|send|recv|prepost|wait|";

        if (size > 1024) {
            iter_count = ITER_COUNT_LARGE;
        }

        buf_size = size*iter_count;
        buf = malloc (buf_size);
        memset(buf, 0, buf_size);

        CUDA_CHECK(hipMalloc((void **)&sbuf_d, buf_size));
        CUDA_CHECK(hipMemset(sbuf_d, 0, buf_size));

        CUDA_CHECK(hipMalloc((void **)&rbuf_d, buf_size));
        CUDA_CHECK(hipMemset(rbuf_d, 0, buf_size));

        MP_CHECK(mp_register(sbuf_d, buf_size, &sreg, 0));
        MP_CHECK(mp_register(rbuf_d, buf_size, &rreg, 0));

        if (!my_rank) {
            if (prof_init(&prof_normal, 1000,  1000, "1us", 100, 1, tags)) {
                fprintf(stderr, "error in prof_init init.\n");
                exit(-1);
            }
            if (prof_init(&prof_async, 1000,  1000, "1us", 100, 1, tags)) {
                fprintf(stderr, "error in prof_init init.\n");
                exit(-1);
            }

            prof_start = 1;
        }

        if (!my_rank) fprintf(stdout, "%10d", size);

        /*warmup*/
        latency = sr_exchange(MPI_COMM_WORLD, size, iter_count, validate, 0/*kernel_time*/, 1/*use_async*/, NULL/*prof*/);
        latency = sr_exchange(MPI_COMM_WORLD, size, iter_count, validate, 1/*kernel_time*/, 1/*use_async*/, NULL/*prof*/);
        latency = sr_exchange(MPI_COMM_WORLD, size, iter_count, validate, 0/*kernel_time*/, 0/*use_async*/, NULL/*prof*/);
        latency = sr_exchange(MPI_COMM_WORLD, size, iter_count, validate, 1/*kernel_time*/, 0/*use_async*/, NULL/*prof*/);

        /*Normal*/
        latency = sr_exchange(MPI_COMM_WORLD, size, iter_count, validate, 0/*kernel_time*/, 0/*use_async*/, NULL/*prof*/);

        kernel_time = (comm_comp_ratio > 0) ? comm_comp_ratio*latency : kernel_time;
        if (!my_rank) fprintf(stdout, "\t   %10d", kernel_time);
        if (!my_rank) fprintf(stdout, "\t   %8.2lf", latency);

        hipProfilerStart();

        /*Normal + Kernel*/
        latency = sr_exchange(MPI_COMM_WORLD, size, iter_count, validate, kernel_time, 0/*use_async*/, &prof_normal/*prof*/);
        if (!my_rank) fprintf(stdout, "\t   %8.2lf", latency);

        hipProfilerStop();

        /*Async*/
        latency = sr_exchange(MPI_COMM_WORLD, size, iter_count, validate, 0/*kernel_time*/, 1/*use_async*/, NULL/*prof*/);
        if (!my_rank) fprintf(stdout, "\t   %8.2lf", latency);

        hipProfilerStart();

        /*Async + Kernel*/
        latency = sr_exchange(MPI_COMM_WORLD, size, iter_count, validate, kernel_time, 1/*use_async*/, &prof_async/*prof*/);
        if (!my_rank) fprintf(stdout, "\t   %8.2lf \n", latency);


        hipProfilerStop();

        if (!my_rank && validate) fprintf(stdout, "SendRecv test passed validation with message size: %d \n", size);

        if (!my_rank) {
            prof_dump(&prof_normal);
            prof_dump(&prof_async);
        }

        mp_deregister(&sreg);
        mp_deregister(&rreg);

        CUDA_CHECK(hipFree(sbuf_d));
        CUDA_CHECK(hipFree(rbuf_d));
        free(buf);
    }

    CUDA_CHECK(hipStreamDestroy(stream));
    free(sreq);
    free(rreq);

    mp_finalize ();

    MPI_Barrier(MPI_COMM_WORLD);

    MPI_Finalize();

    return 0;
}
