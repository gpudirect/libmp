#include "hip/hip_runtime.h"
/****
 * Copyright (c) 2011-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ****/

#include <string.h>
#include <stdio.h>
#include <assert.h>
#include <limits.h>
#include <sys/types.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <mpi.h>
#include <gdsync.h>
#include <mp.h>

#include "prof.h"

#define CUDA_CHECK(stmt)                                \
do {                                                    \
    hipError_t result = (stmt);                        \
    if (hipSuccess != result) {                        \
        fprintf(stderr, "[%s:%d] cuda failed with %s \n",   \
         __FILE__, __LINE__,hipGetErrorString(result));\
        exit(-1);                                       \
    }                                                   \
    assert(hipSuccess == result);                      \
} while (0)

#define CU_CHECK(stmt)                                 \
do {                                                    \
    hipError_t result = (stmt);                           \
    if (hipSuccess != result) {                        \
        fprintf(stderr, "[%s:%d] cuda failed with %d \n",   \
         __FILE__, __LINE__, result);\
        exit(-1);                                       \
    }                                                   \
    assert(hipSuccess == result);                     \
} while (0)

#define CUDA_LAST_ERROR()                                 \
do {                                                    \
    hipError_t result = hipGetLastError();                            \
    if (hipSuccess != result) {                        \
        fprintf(stderr, "[%s:%d] cuda failed with %s \n",   \
         __FILE__, __LINE__,hipGetErrorString(result));\
        exit(-1);                                       \
    }                                                   \
    assert(hipSuccess == result);                      \
} while (0)


#define MP_CHECK(stmt)                                  \
do {                                                    \
    int result = (stmt);                                \
    if (0 != result) {                                  \
        fprintf(stderr, "[%s:%d] mp call failed \n",    \
         __FILE__, __LINE__);                           \
        exit(-1);                                       \
    }                                                   \
    assert(0 == result);                                \
} while (0)

int enable_debug_prints = 0;
#define mp_dbg_msg(FMT, ARGS...)  do                                    \
{                                                                       \
    if (enable_debug_prints)  {                                              \
        fprintf(stderr, "[%d] [%d] MP DBG  %s() " FMT, getpid(),  my_rank, __FUNCTION__ , ## ARGS); \
        fflush(stderr);                                                 \
    }                                                                   \
} while(0)

#define MAX_SIZE 1*1024*1024 
#define ITER_COUNT_SMALL 1000
#define ITER_COUNT_LARGE 1000

struct prof prof_normal;
struct prof prof_async;
struct prof prof_mpi;
int prof_start = 0;
int prof_idx = 0;

static const int over_sub_factor = 2;
int gpu_num_sm;
int enable_ud = 0;
int gpu_id = -1;

int comm_size, my_rank, peer;
int steps_per_batch = 20, batches_inflight = 4;
int enable_async = 1;
int calc_size = 128*1024;
int use_calc_size = 1;
volatile uint32_t tracking_event = 0;
int use_gpu_buffers=0;

__device__ int counter;
__device__ int clockrate;

__global__ void calc_kernel(int n, float c, float *in, float *out)
{
        const uint tid = threadIdx.x;
        const uint bid = blockIdx.x;
        const uint block_size = blockDim.x;
        const uint grid_size = gridDim.x;
        const uint gid = tid + bid*block_size;
        const uint n_threads = block_size*grid_size;
        for (int i=gid; i<n; i += n_threads)
                out[i] = in[i] * c;
}

int gpu_launch_calc_kernel(size_t size, hipStream_t stream)
{
        const int nblocks = over_sub_factor * gpu_num_sm;
        const int nthreads = 32*2;
        int n = size / sizeof(float);
        static float *in = NULL;
        static float *out = NULL;
        if (!in) {
                CUDA_CHECK(hipMalloc((void **)&in, size));
                CUDA_CHECK(hipMalloc((void **)&out, size));

                CUDA_CHECK(hipMemset((void *)in, 1, size));
                CUDA_CHECK(hipMemset((void *)out, 1, size));
        }
        calc_kernel<<<nblocks, nthreads, 0, stream>>>(n, 1.0f, in, out);
        CUDA_CHECK(hipGetLastError());
        return 0;
}

__global__ void dummy_kernel(double time)
{
    long long int start, stop;
    double usec;

    start = clock64();
    do {
        stop = clock64();
	usec = ((double)(stop-start)*1000)/((double)clockrate); 
	counter = usec;
    } while(usec < time);
}

/*application and pack buffers*/
void *buf = NULL, *sbuf_d = NULL, *rbuf_d = NULL;
hipStream_t stream;
size_t buf_size; 

/*mp specific objects*/
mp_request_t *sreq = NULL;
mp_request_t *rreq = NULL;
mp_reg_t sreg, rreg; 
double time_start, time_stop;
MPI_Request * sreq_mpi;
MPI_Request * rreq_mpi;

int batch_to_rreq_idx (int batch_idx) { 
     return (batch_idx % (batches_inflight + 1))*steps_per_batch;
}

int batch_to_sreq_idx (int batch_idx) { 
     return (batch_idx % batches_inflight)*steps_per_batch;
}

void post_recv (int size, int batch_index)
{
    int j;
    int req_idx = batch_to_rreq_idx (batch_index);
 
    for (j=0; j<steps_per_batch; j++) {
        MP_CHECK(mp_irecv ((void *)((uintptr_t)rbuf_d), size, peer, &rreg, &rreq[req_idx + j]));
    }
}

void post_recv_mpi (int size, int batch_index)
{
    int j;
    int req_idx = batch_to_rreq_idx (batch_index);
 
    for (j=0; j<steps_per_batch; j++) {
        MP_CHECK(MPI_Irecv ((void *)((uintptr_t)rbuf_d), size, MPI_CHAR, peer, my_rank, MPI_COMM_WORLD, &rreq_mpi[req_idx + j]));
    }
}


void wait_send (int batch_index) 
{
    int j;
    int req_idx = batch_to_sreq_idx (batch_index); 

    for (j=0; j<steps_per_batch; j++) {
        MP_CHECK(mp_wait(&sreq[req_idx + j]));
    }
}

void wait_send_mpi (int batch_index) 
{
    int j;
    int req_idx = batch_to_sreq_idx (batch_index); 

    for (j=0; j<steps_per_batch; j++) {
        MP_CHECK(MPI_Wait(&sreq_mpi[req_idx + j], MPI_STATUS_IGNORE));
    }
}


void wait_recv (int batch_index) 
{
    int j;
    int req_idx = batch_to_rreq_idx (batch_index);
 
    for (j=0; j<steps_per_batch; j++) {
        MP_CHECK(mp_wait(&rreq[req_idx + j]));
    }
}

void wait_recv_mpi (int batch_index) 
{
    int j;
    int req_idx = batch_to_rreq_idx (batch_index);
 
    for (j=0; j<steps_per_batch; j++) {
        MP_CHECK(MPI_Wait(&rreq_mpi[req_idx + j], MPI_STATUS_IGNORE));
    }
}

void post_work_async (int size, int batch_index, double kernel_size) 
{
    int j;
    int sreq_idx = batch_to_sreq_idx (batch_index);
    int rreq_idx = batch_to_rreq_idx (batch_index);
   
    for (j=0; j<steps_per_batch; j++) {
	if (!my_rank) { 
            MP_CHECK(mp_wait_on_stream(&rreq[rreq_idx + j], stream));

            if (kernel_size > 0) {
                if (use_calc_size > 0)
                   gpu_launch_calc_kernel(kernel_size, stream);
                else
                   dummy_kernel <<<1, 1, 0, stream>>> (kernel_size);
            }


            MP_CHECK(mp_isend_on_stream ((void *)((uintptr_t)sbuf_d), size, peer, &sreg, &sreq[sreq_idx + j], stream));
	} else {
            MP_CHECK(mp_isend_on_stream ((void *)((uintptr_t)sbuf_d), size, peer, &sreg, &sreq[sreq_idx + j], stream));

            MP_CHECK(mp_wait_on_stream(&rreq[rreq_idx + j], stream));

            if (kernel_size > 0) {
                if (use_calc_size > 0)
                   gpu_launch_calc_kernel(kernel_size, stream);
                else
                   dummy_kernel <<<1, 1, 0, stream>>> (kernel_size);
            }
	}
    }
}

void post_work_sync (int size, int batch_index, double kernel_size) 
{
    int j;
    int rreq_idx = batch_to_rreq_idx (batch_index);
    int sreq_idx = batch_to_sreq_idx (batch_index);

    for (j=0; j<steps_per_batch; j++) {
	if (!my_rank) { 
            MP_CHECK(mp_wait(&rreq[rreq_idx + j]));

            if (kernel_size > 0) {
                if (use_calc_size > 0)
                   gpu_launch_calc_kernel(kernel_size, stream);
                else
                   dummy_kernel <<<1, 1, 0, stream>>> (kernel_size);
                CUDA_CHECK(hipStreamSynchronize(stream));
            }

            MP_CHECK(mp_isend ((void *)((uintptr_t)sbuf_d), size, peer, &sreg, &sreq[sreq_idx + j]));
	} else {
            MP_CHECK(mp_isend ((void *)((uintptr_t)sbuf_d), size, peer, &sreg, &sreq[sreq_idx + j]));

            MP_CHECK(mp_wait(&rreq[rreq_idx + j]));

            if (kernel_size > 0) {
                if (use_calc_size > 0)
                   gpu_launch_calc_kernel(kernel_size, stream);
                else
                   dummy_kernel <<<1, 1, 0, stream>>> (kernel_size);
                CUDA_CHECK(hipStreamSynchronize(stream));
            }
        }
    }
}

void post_work_mpi (int size, int batch_index, double kernel_size) 
{
    int j;
    int rreq_idx = batch_to_rreq_idx (batch_index);
    int sreq_idx = batch_to_sreq_idx (batch_index);

    for (j=0; j<steps_per_batch; j++) {
    if (!my_rank) { 
            MP_CHECK(MPI_Wait(&rreq_mpi[rreq_idx + j], MPI_STATUS_IGNORE));

            if (kernel_size > 0) {
                if (use_calc_size > 0)
                   gpu_launch_calc_kernel(kernel_size, stream);
                else
                   dummy_kernel <<<1, 1, 0, stream>>> (kernel_size);
                CUDA_CHECK(hipStreamSynchronize(stream));
            }
            MPI_Isend((void *)(uintptr_t)sbuf_d, size, MPI_CHAR, peer, peer, MPI_COMM_WORLD, &sreq_mpi[sreq_idx + j]);
    } else {
            MPI_Isend((void *)(uintptr_t)sbuf_d, size, MPI_CHAR, peer, peer, MPI_COMM_WORLD, &sreq_mpi[sreq_idx + j]);

            MP_CHECK(MPI_Wait(&rreq_mpi[rreq_idx + j], MPI_STATUS_IGNORE));

            if (kernel_size > 0) {
                if (use_calc_size > 0)
                   gpu_launch_calc_kernel(kernel_size, stream);
                else
                   dummy_kernel <<<1, 1, 0, stream>>> (kernel_size);
                CUDA_CHECK(hipStreamSynchronize(stream));
            }
        }
    }
}

double prepost_latency;

double sr_exchange (MPI_Comm comm, int size, int iter_count, double kernel_size, int use_async)
{
    int j;
    double latency;
    double time_start, time_stop;
    int batch_count, wait_send_batch = 0, wait_recv_batch = 0;
    struct prof *prof = NULL;

    prof = (use_async) ? &prof_async : &prof_normal;
 
    assert((iter_count%steps_per_batch) == 0);
    batch_count = iter_count/steps_per_batch;
    tracking_event = 0;

    post_recv (size, 0);

    MPI_Barrier(MPI_COMM_WORLD);

    time_start = MPI_Wtime();

    for (j=0; (j<batches_inflight) && (j<batch_count); j++) { 
        if (j<(batch_count-1)) {
            post_recv (size, j+1);
        }

        if (use_async) { 
            post_work_async (size, j, kernel_size);
        } else {               
            post_work_sync (size, j, kernel_size);
	   }
    }

    time_stop = MPI_Wtime();

    prepost_latency = ((time_stop - time_start)*1e6);
    
    time_start = MPI_Wtime();

    wait_send_batch = wait_recv_batch = 0;
    prof_idx = 0;
    while (wait_send_batch < batch_count) { 
        if (!my_rank && prof_start) PROF(prof, prof_idx++);

    	if (use_async) {
    	    wait_recv (wait_recv_batch);
            wait_recv_batch++;
    	}

        if (!my_rank && prof_start) PROF(prof, prof_idx++); 

        wait_send (wait_send_batch);
        wait_send_batch++;

        if (!my_rank && prof_start) PROF(prof, prof_idx++);

        if (j < (batch_count-1)) {
            post_recv (size, j+1);
        }

        if (!my_rank && prof_start) PROF(prof, prof_idx++);

        if (j < batch_count) { 
            if (use_async) { 
                    post_work_async (size, j, kernel_size);
            } else {
                    post_work_sync (size, j, kernel_size);
            }
        }

        if (!my_rank && prof_start)  {
            PROF(prof, prof_idx++);
            prof_update(prof);
            prof_idx = 0;
        }

	   j++;
    }

    MPI_Barrier(comm);

    time_stop = MPI_Wtime();
    latency = (((time_stop - time_start)*1e6 + prepost_latency)/(iter_count));

    CUDA_CHECK(hipDeviceSynchronize());

    return latency;
}

double sr_exchange_MPI (MPI_Comm comm, int size, int iter_count, double kernel_size)
{
    int j;
    double latency;
    double time_start, time_stop;
    int batch_count, wait_send_batch = 0, wait_recv_batch = 0;
    struct prof *prof = NULL;

    prof = &prof_mpi;
 
    assert((iter_count%steps_per_batch) == 0);
    batch_count = iter_count/steps_per_batch;
    tracking_event = 0;
    
    post_recv_mpi (size, 0);

    MPI_Barrier(MPI_COMM_WORLD);

    time_start = MPI_Wtime();

    for (j=0; (j<batches_inflight) && (j<batch_count); j++) { 
        if (j<(batch_count-1)) {
            post_recv_mpi (size, j+1);
        }

        post_work_mpi (size, j, kernel_size);
    }

    time_stop = MPI_Wtime();

    prepost_latency = ((time_stop - time_start)*1e6);
    
    time_start = MPI_Wtime();

    wait_send_batch = wait_recv_batch = 0;
    prof_idx = 0;
    while (wait_send_batch < batch_count) 
    { 

        if (!my_rank && prof_start) PROF(prof, prof_idx++);

        //if (!my_rank && prof_start) PROF(prof, prof_idx++); 

        wait_send_mpi (wait_send_batch);
        wait_send_batch++;

        if (!my_rank && prof_start) PROF(prof, prof_idx++);

        if (j < (batch_count-1)) {
            post_recv_mpi (size, j+1);
        }

        if (!my_rank && prof_start) PROF(prof, prof_idx++);


        if (j < batch_count) { 
            post_work_mpi (size, j, kernel_size);
        }

        if (!my_rank && prof_start)  {
            PROF(prof, prof_idx++);
            prof_update(prof);
            prof_idx = 0;
        }

        j++;
    }

    MPI_Barrier(comm);

    time_stop = MPI_Wtime();
    latency = (((time_stop - time_start)*1e6 + prepost_latency)/(iter_count));

    CUDA_CHECK(hipDeviceSynchronize());

    return latency;
}


int main (int argc, char *argv[])
{
    int iter_count, max_size, size, dev_count, local_rank, dev_id = 0;
    int kernel_size = 20;
    int comm_comp_ratio = 0;
    int validate = 0;

    size = 1;
    max_size = MAX_SIZE;

    char *value = getenv("ENABLE_VALIDATION");
    if (value != NULL) {
	validate = atoi(value);
    }
 
    value = getenv("ENABLE_DEBUG_MSG");
    if (value != NULL) {
	enable_debug_prints = atoi(value);
    }

    value = getenv("KERNEL_TIME");
    if (value != NULL) {
	kernel_size = atoi(value);
    }

    value = getenv("COMM_COMP_RATIO");
    if (value != NULL) {
        comm_comp_ratio = atoi(value);
    }

    value = getenv("CALC_SIZE");
    if (value != NULL) {
        calc_size = atoi(value);
    }

    use_calc_size = 1;
    value = getenv("USE_CALC_SIZE");
    if (value != NULL) {
        use_calc_size = atoi(value);
    }

    value = getenv("STEPS_PER_BATCH");
    if (value != NULL) {
        steps_per_batch = atoi(value);
    }

    value = getenv("BATCHES_INFLIGHT");
    if (value != NULL) {
        batches_inflight = atoi(value);
    }

    value = getenv("SIZE");
    if (value != NULL && atoi(value)) {
        size = atoi(value);
    }

    value = getenv("MP_ENABLE_UD");
    if (value != NULL) {
        enable_ud = atoi(value);
    }

    if (enable_ud) {
	if (max_size > 4096) { 
	    max_size = 4096;
        }
    }

    value = getenv("USE_GPU_BUFFERS");
    if (value != NULL) {
        use_gpu_buffers = atoi(value);
    }

    printf("use_gpu_buffers=%d\n", use_gpu_buffers);

    while(1) {
        int c;
        c = getopt(argc, argv, "d:h");
        if (c == -1)
            break;

        switch(c) {
        case 'd':
            gpu_id = strtol(optarg, NULL, 0);
            break;
	case 'h':
            printf("syntax: %s [-d <gpu_id]\n", argv[0]);
	    break;
        default:
            printf("ERROR: invalid option\n");
            exit(EXIT_FAILURE);
        }
    }

    char *tags = "wait_recv|wait_send|post_recv|post_work";

    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &comm_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &my_rank);

    if (comm_size != 2) { 
	fprintf(stderr, "this test requires exactly two processes \n");
        exit(-1);
    }

    CUDA_CHECK(hipGetDeviceCount(&dev_count));
    if (dev_count <= 0) {
        fprintf(stderr, "no CUDA devices found \n");
        exit(-1);
    }

    if (getenv("MV2_COMM_WORLD_LOCAL_RANK") != NULL) {
        local_rank = atoi(getenv("MV2_COMM_WORLD_LOCAL_RANK"));
    } else if (getenv("OMPI_COMM_WORLD_LOCAL_RANK") != NULL) {
        local_rank = atoi(getenv("OMPI_COMM_WORLD_LOCAL_RANK"));
    } else {
        local_rank = 0;
    }

    if (gpu_id >= 0) {
        dev_id = gpu_id;
    } else if (getenv("USE_GPU")) {
        dev_id = atoi(getenv("USE_GPU"));
    } else {
        dev_id = local_rank%dev_count;
    }
    if (dev_id >= dev_count) {
        fprintf(stderr, "invalid dev_id\n");
        exit(-1);
    }

    fprintf(stdout, "[%d] local_rank: %d dev_count: %d using GPU device: %d\n", my_rank, local_rank, dev_count, dev_id);

    CUDA_CHECK(hipSetDevice(dev_id));
    CUDA_CHECK(hipFree(0));

    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, dev_id));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(clockrate), (void *)&prop.clockRate, sizeof(int), 0, hipMemcpyHostToDevice));
    gpu_num_sm = prop.multiProcessorCount;

    fprintf(stdout, "[%d] GPU %d: %s PCIe %d:%d:%d\n", my_rank, dev_id, prop.name, prop.pciDomainID, prop.pciBusID, prop.pciDeviceID);

    peer = !my_rank;
    MP_CHECK(mp_init (MPI_COMM_WORLD, &peer, 1, MP_INIT_DEFAULT, dev_id));

    iter_count = ITER_COUNT_SMALL;
    if (!my_rank) { 
        fprintf(stdout, "steps_per_batch: %d batches_inflight: %d \n", 
   		steps_per_batch, batches_inflight);
        fprintf(stdout, "WARNING: dumping round-trip latency!!!\n");
    }

    /*allocating requests*/
    sreq = (mp_request_t *) malloc(steps_per_batch*batches_inflight*sizeof(mp_request_t));
    rreq = (mp_request_t *) malloc(steps_per_batch*(batches_inflight + 1)*sizeof(mp_request_t));

    sreq_mpi = (MPI_Request *) malloc(steps_per_batch*batches_inflight*sizeof(MPI_Request));
    rreq_mpi = (MPI_Request *) malloc(steps_per_batch*(batches_inflight + 1)*sizeof(MPI_Request));
   
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, 0));	

    if (!my_rank) {   
    	if (use_calc_size) { 
    		fprintf(stdout, "%10s \t %10s \t %10s \t %10s \t  %10s \t %10s \t %10s \t %10s\n", "Size", "CalcSize", "No-async", "No-async+Kern", "Async", "Async+Kern", "MPI", "MPI+Kern");
    	} else {
    		fprintf(stdout, "%10s \t %10s \t  %10s \t %10s \t %10s \t  %10s \t %10s \t %10s\n", "Size", "KernelTime", "No-async", "No-async+Kern", "Async", "Async+Kern", "MPI", "MPI+Kern");
    	}
    }

    if (size != 1) size = max_size = size;
    for (; size<=max_size; size*=2) 
    {
        double latency;

        if (size > 1024) {
            iter_count = ITER_COUNT_LARGE;
        }

        buf_size = size;

        buf = malloc (buf_size);
        memset(buf, 0, buf_size); 
        if(use_gpu_buffers == 0)
        {
            CUDA_CHECK(hipHostMalloc((void **)&sbuf_d, buf_size));
            memset(sbuf_d, 0, buf_size);

            CUDA_CHECK(hipHostMalloc((void **)&rbuf_d, buf_size));
            memset(rbuf_d, 0, buf_size);   
        }
        else
        {
            CUDA_CHECK(hipMalloc((void **)&sbuf_d, buf_size));
            CUDA_CHECK(hipMemset(sbuf_d, 0, buf_size)); 

            CUDA_CHECK(hipMalloc((void **)&rbuf_d, buf_size));
            CUDA_CHECK(hipMemset(rbuf_d, 0, buf_size)); 
        }

        MP_CHECK(mp_register(sbuf_d, buf_size, &sreg, 0));
        MP_CHECK(mp_register(rbuf_d, buf_size, &rreg, 0));

        if (!my_rank) fprintf(stdout, "%10d", size);

#if 0
        if (!my_rank) fprintf(stdout, "sleeping 10s\n");
        sleep(10);
        MPI_Barrier(MPI_COMM_WORLD);
#endif

        /*warmup*/
        latency = sr_exchange(MPI_COMM_WORLD, size, iter_count, 0/*kernel_size*/, 1/*use_async*/);

        MPI_Barrier(MPI_COMM_WORLD);

        latency = sr_exchange(MPI_COMM_WORLD, size, iter_count, 0/*kernel_size*/, 0/*use_async*/);

        MPI_Barrier(MPI_COMM_WORLD);

        latency = sr_exchange_MPI(MPI_COMM_WORLD, size, iter_count, 0/*kernel_size*/);
    
        MPI_Barrier(MPI_COMM_WORLD);

	/*Normal*/
        latency = sr_exchange(MPI_COMM_WORLD, size, iter_count, 0/*kernel_size*/, 0/*use_async*/);

        MPI_Barrier(MPI_COMM_WORLD);
     
     
        if (use_calc_size) kernel_size = calc_size; 
        else  kernel_size = (comm_comp_ratio > 0) ? comm_comp_ratio*(latency/2) : kernel_size;

        if (!my_rank) fprintf(stdout, "\t   %10d", kernel_size);
        if (!my_rank) fprintf(stdout, "\t   %8.2lf", latency, prepost_latency);
        //if (!my_rank) fprintf(stdout, "\t   %8.2lf (%8.2lf)", latency, prepost_latency);

        hipProfilerStart();
        if (!my_rank) { 
            if (prof_init(&prof_normal, 10000, 10000, "10us", 100, 1, tags)) {
                    fprintf(stderr, "error in prof_init init.\n");
                    exit(-1);
                }
                if (prof_init(&prof_async, 10000, 10000, "10us", 100, 1, tags)) {
                    fprintf(stderr, "error in prof_init init.\n");
                    exit(-1);
                }
                if (prof_init(&prof_mpi, 10000,  10000, "10us", 100, 1, tags)) {
                    fprintf(stderr, "error in prof_init init.\n");
                    exit(-1);
                }
                prof_start = 1;
        }

	   /*Normal + Kernel*/
        latency = sr_exchange(MPI_COMM_WORLD, size, iter_count, kernel_size, 0/*use_async*/);

        MPI_Barrier(MPI_COMM_WORLD);

        if (!my_rank) fprintf(stdout, "\t   %8.2lf ", latency /*, prepost_latency */);
        //if (!my_rank) fprintf(stdout, "\t   %8.2lf (%8.2lf)", latency, prepost_latency);

        if (!my_rank) {
            prof_start = 0;
        }
        hipProfilerStop();

	/*Async*/
        latency = sr_exchange(MPI_COMM_WORLD, size, iter_count, 0/*kernel_size*/, 1/*use_async*/);

        MPI_Barrier(MPI_COMM_WORLD);
 
        if (!my_rank) fprintf(stdout, "\t   %8.2lf ", latency /*, prepost_latency */);
        //if (!my_rank) fprintf(stdout, "\t   %8.2lf (%8.2lf)", latency, prepost_latency);

        hipProfilerStart();
        if (!my_rank) {
            prof_start = 1;
        }

	/*Async + Kernel*/
        latency = sr_exchange(MPI_COMM_WORLD, size, iter_count, kernel_size, 1/*use_async*/);

        MPI_Barrier(MPI_COMM_WORLD);

        if (!my_rank) fprintf(stdout, "\t   %8.2lf ", latency /*, prepost_latency */);
        //if (!my_rank) fprintf(stdout, "\t   %8.2lf (%8.2lf) \n", latency, prepost_latency);

        if (!my_rank) {
            prof_start = 0;
        }
        hipProfilerStop();


/*MPI*/
        latency = sr_exchange_MPI(MPI_COMM_WORLD, size, iter_count, 0/*kernel_size*/);

        MPI_Barrier(MPI_COMM_WORLD);
 
        if (!my_rank) fprintf(stdout, "\t   %8.2lf ", latency /*, prepost_latency */);
        //if (!my_rank) fprintf(stdout, "\t   %8.2lf (%8.2lf)", latency, prepost_latency);

        hipProfilerStart();
        if (!my_rank) {
            prof_start = 1;
        }

    /*MPI + Kernel*/
        latency = sr_exchange_MPI(MPI_COMM_WORLD, size, iter_count, kernel_size);

        MPI_Barrier(MPI_COMM_WORLD);

        if (!my_rank) fprintf(stdout, "\t   %8.2lf \n", latency /*, prepost_latency */);
        //if (!my_rank) fprintf(stdout, "\t   %8.2lf (%8.2lf) \n", latency, prepost_latency);

        prof_start = 0;
        hipProfilerStop();

        if (!my_rank && validate) fprintf(stdout, "SendRecv test passed validation with message size: %d \n", size);

        if (!my_rank) {
	       prof_dump(&prof_normal);
	       prof_dump(&prof_async);
           prof_dump(&prof_mpi);
        }

        mp_deregister(&sreg);
        mp_deregister(&rreg);

        if(use_gpu_buffers == 0)
        {
            CUDA_CHECK(hipHostFree(sbuf_d));
            CUDA_CHECK(hipHostFree(rbuf_d));
        }
        else
        {
            CUDA_CHECK(hipFree(sbuf_d));
            CUDA_CHECK(hipFree(rbuf_d));            
        }
        free(buf);
    }

    CUDA_CHECK(hipStreamDestroy(stream));
    free(sreq);
    free(rreq);

    mp_finalize ();

    MPI_Barrier(MPI_COMM_WORLD);

    MPI_Finalize();

    return 0;
}
